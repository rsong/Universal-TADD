#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#define IMUL(a, b) __mul24(a, b)

////////////////////////////////////////////////////////////////////////////////
// Padding kernels
////////////////////////////////////////////////////////////////////////////////
typedef float2 Complex;


////////////////////////////////////////////////////////////////////////////////
// Helper functions
////////////////////////////////////////////////////////////////////////////////
//Round a / b to nearest higher integer value
__host__  int iDivUp(int a, int b){
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Align a to nearest higher multiple of b
__host__  int iAlignUp(int a, int b){
	return (a % b != 0) ?  (a - a % b + b) : a;
}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
__host__ int calculateFFTsize(int dataSize){
	//Highest non-zero bit position of dataSize
	int hiBit;
	//Neares lower and higher powers of two numbers for dataSize
	unsigned int lowPOT, hiPOT;

	//Align data size to a multiple of half-warp
	//in order to have each line starting at properly aligned addresses
	//for coalesced global memory writes in padKernel() and padData()
	dataSize = iAlignUp(dataSize, 16);

	//Find highest non-zero bit
	for(hiBit = 31; hiBit >= 0; hiBit--)
		if(dataSize & (1U << hiBit)) break;

	//No need to align, if already power of two
	lowPOT = 1U << hiBit;
	if(lowPOT == dataSize) return dataSize;

	//Align to a nearest higher power of two, if the size is small enough,
	//else align only to a nearest higher multiple of 512,
	//in order to save computation and memory bandwidth
	hiPOT = 1U << (hiBit + 1);
	//if(hiPOT <= 1024)
		return hiPOT;
	//else 
	//	return iAlignUp(dataSize, 512);
}

////////////////////////////////////////////////////////////////////////////////
// Cyclically shift convolution kernel, so that the center is at (0, 0)
////////////////////////////////////////////////////////////////////////////////
texture<float, 2, hipReadModeElementType> texKernel;

__global__ void padKernel(
	float *d_PaddedKernel,
	int fftW,
	int fftH,
	int kernelW,
	int kernelH,
	int kernelX,
	int kernelY
){
	const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;

	if(x < kernelW && y < kernelH){
		int kx = x - kernelX; if(kx < 0) kx += fftW;
		int ky = y - kernelY; if(ky < 0) ky += fftH;
		d_PaddedKernel[IMUL(ky, fftW) + kx] = tex2D(texKernel, (float)x + 0.5f, (float)y + 0.5f);
	}
}


////////////////////////////////////////////////////////////////////////////////
// Copy input data array to the upper left corner and pad by border values
////////////////////////////////////////////////////////////////////////////////
texture<float, 2, hipReadModeElementType> texData;

__global__ void padData(
	float *d_PaddedData,
	int fftW,
	int fftH,
	int dataW,
	int dataH,
	int kernelW,
	int kernelH,
	int kernelX,
	int kernelY
){
	const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	const int borderW = dataW + kernelX;
	const int borderH = dataH + kernelY;
	int dx;
	int dy;

	if(x < fftW && y < fftH){
		if(x < dataW) dx = x;
		if(y < dataH) dy = y;
		if(x >= dataW && x < borderW) dx = dataW - 1;
		if(y >= dataH && y < borderH) dy = dataH - 1;
		if(x >= borderW) dx = 0;
		if(y >= borderH) dy = 0;

		d_PaddedData[IMUL(y, fftW) + x] =
			tex2D(texData, (float)dx + 0.5f, (float)dy + 0.5f);
	}
}


////////////////////////////////////////////////////////////////////////////////
// Modulate Fourier image of padded data by Fourier image of padded kernel
// and normalize by FFT size
////////////////////////////////////////////////////////////////////////////////
__device__ void complexMulAndScale(Complex& a, Complex b, float c){
	Complex t = {c * (a.x * b.x - a.y * b.y), c * (a.y * b.x + a.x * b.y)};
	a = t;
}

__global__ void modulateAndNormalize(
	Complex *fft_PaddedData,
	Complex *fft_PaddedKernel,
	int dataN
){
	const int     tid = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int threadN = IMUL(blockDim.x, gridDim.x);
	const float     q = 1.0f / (float)dataN;

	for(int i = tid; i < dataN; i += threadN)
		complexMulAndScale(fft_PaddedData[i], fft_PaddedKernel[i], q);
}

////////////////////////////////////////////////////////////////////////////////
// FFT convolution program
////////////////////////////////////////////////////////////////////////////////
__host__ void fftFunction(float* output, float *in_Data, float *in_Kernel, int DATA_H, int DATA_W, int KERNEL_H, int KERNEL_W ) {

	hipArray *a_Kernel, *a_Data;
	hipChannelFormatDesc float2tex = hipCreateChannelDesc<float>();
	float *d_PaddedKernel, *d_PaddedData;
	Complex *fft_PaddedKernel, *fft_PaddedData;

	hipfftHandle FFTplan_R2C;
	hipfftHandle FFTplan_C2R;

	int KERNEL_X, KERNEL_Y, PADDING_W, PADDING_H, FFT_W, FFT_H, FFT_SIZE, KERNEL_SIZE, DATA_SIZE, CFFT_SIZE;

	// we expect 2 inputs: prhs[0] -- data, prhs[1] -- kernel
	// Kernel center position
	KERNEL_X = KERNEL_W/2;
	KERNEL_Y = KERNEL_H/2;

	// Width and height of padding for "clamp to border" addressing mode
	PADDING_W = KERNEL_W - 1;
	PADDING_H = KERNEL_H - 1;

	// Derive FFT size from data and kernel dimensions
	FFT_W = calculateFFTsize(DATA_W + PADDING_W);
	FFT_H = calculateFFTsize(DATA_H + PADDING_H);


	//fprintf(stderr,"Calculating byte sizes..\n");
	FFT_SIZE = FFT_W * FFT_H * sizeof(float);
	CFFT_SIZE = FFT_W * FFT_H * sizeof(Complex);
	KERNEL_SIZE = KERNEL_W * KERNEL_H * sizeof(float);
	DATA_SIZE = DATA_W * DATA_H * sizeof(float);

	hipMallocArray(&a_Kernel, &float2tex, KERNEL_W, KERNEL_H) ;
	hipMallocArray(&a_Data,   &float2tex,   DATA_W,   DATA_H) ;
	hipMalloc((void **)&d_PaddedKernel, 	FFT_SIZE) ;
	hipMalloc((void **)&d_PaddedData,   	FFT_SIZE) ;
	hipMalloc((void **)&fft_PaddedKernel, 	CFFT_SIZE);
	hipMalloc((void **)&fft_PaddedData, 	CFFT_SIZE);

	hipfftPlan2d(&FFTplan_C2R, FFT_H, FFT_W, HIPFFT_C2R) ;
	hipfftPlan2d(&FFTplan_R2C, FFT_H, FFT_W, HIPFFT_R2C) ;

	hipMemset(d_PaddedKernel, 0, FFT_SIZE) ;
	hipMemset(d_PaddedData,   0, FFT_SIZE) ;

	// copying input data and convolution kernel from host to CUDA arrays
	hipMemcpyToArray(a_Kernel, 0, 0, in_Kernel, KERNEL_SIZE, hipMemcpyHostToDevice) ;
	hipMemcpyToArray(a_Data,   0, 0, in_Data,   DATA_SIZE,   hipMemcpyHostToDevice) ;
	//binding CUDA arrays to texture references
	hipBindTextureToArray(texKernel, a_Kernel) ;
	hipBindTextureToArray(texData,   a_Data)   ;

	//Block width should be a multiple of maximum coalesced write size 
	//for coalesced memory writes in padKernel() and padData()
	dim3 threadBlock(16, 12);
	dim3 kernelBlockGrid(iDivUp(KERNEL_W, threadBlock.x), iDivUp(KERNEL_H, threadBlock.y));
	dim3 dataBlockGrid(iDivUp(FFT_W, threadBlock.x), iDivUp(FFT_H, threadBlock.y));

	//fprintf(stderr,"Padding convolution kernel\n");
	// padding convolution kernel
	padKernel<<<kernelBlockGrid, threadBlock>>>(
		d_PaddedKernel,
		FFT_W,
		FFT_H,
		KERNEL_W,
		KERNEL_H,
		KERNEL_X,
		KERNEL_Y
		);

	//fprintf(stderr,"Padding input data array\n");
	// padding input data array
	padData<<<dataBlockGrid, threadBlock>>>(
		d_PaddedData,
		FFT_W,
		FFT_H,
		DATA_W,
		DATA_H,
		KERNEL_W,
		KERNEL_H,
		KERNEL_X,
		KERNEL_Y
		);

	hipfftExecR2C(FFTplan_R2C, (hipfftReal *)d_PaddedKernel, (hipfftComplex *)fft_PaddedKernel);
	hipfftExecR2C(FFTplan_R2C, (hipfftReal *)d_PaddedData, (hipfftComplex *)fft_PaddedData);

	modulateAndNormalize<<<16, 128>>>(
		fft_PaddedData,
		fft_PaddedKernel,
		FFT_W * FFT_H
		);
	hipfftExecC2R(FFTplan_C2R, (hipfftComplex *)fft_PaddedData, (hipfftReal *)d_PaddedData);

	//fprintf(stderr,"Fetching result from GPU\n");
	hipMemcpy(output, d_PaddedData, FFT_SIZE, hipMemcpyDeviceToHost) ;

	//fprintf(stderr,"Freeing memory\n");
	hipUnbindTexture(texData);
	hipUnbindTexture(texKernel);
	hipfftDestroy(FFTplan_C2R);
	hipfftDestroy(FFTplan_R2C);
	hipFree(d_PaddedData);
	hipFree(d_PaddedKernel);
	hipFree(fft_PaddedData);
	hipFree(fft_PaddedKernel);
	hipFreeArray(a_Data);
	hipFreeArray(a_Kernel);

}

float* runFFT(float* DATA, int DATA_W, int DATA_H, float* KERNEL, int KERNEL_W, int KERNEL_H){
	int FFT_W, FFT_H, PADDING_H, PADDING_W;

	// Width and height of padding for "clamp to border" addressing mode
		PADDING_W = KERNEL_W - 1;
		PADDING_H = KERNEL_H - 1;

	// Derive FFT size from data and kernel dimensions
		FFT_W = calculateFFTsize(DATA_W + PADDING_W);
		FFT_H = calculateFFTsize(DATA_H + PADDING_H);
	
   //create output
		float* output=(float*)malloc(DATA_W*DATA_H*1*sizeof(float));

	// If there's just too much data to do in a single run, we need to break it up, eh?
		// how much "too big" is it?
		int MAX_FFT_W = 1048576/FFT_H;

		if ( FFT_W > MAX_FFT_W ) { // we need to break up the data
			
			int STRIP_W = MAX_FFT_W-KERNEL_W+1;
			int STRIP_SIZE = STRIP_W * DATA_H * sizeof(float);
			int OVERLAP_SIZE = DATA_H * KERNEL_W/2 * sizeof(float);
			
			//malloc input and output strips
			float* strip_output=(float*)malloc(STRIP_SIZE);
			float* strip_input=(float*)malloc(STRIP_SIZE);
			
			int REMAIN_W = DATA_W; // counter showing how much of the data remains to be processed

			// Do the first strip
			hipMemcpy(strip_input, DATA, STRIP_SIZE, hipMemcpyHostToHost);
			fftFunction(strip_output, strip_input, KERNEL, DATA_H, STRIP_W, KERNEL_H, KERNEL_W);
			hipMemcpy(output, strip_output, STRIP_SIZE, hipMemcpyHostToHost);
			
			REMAIN_W -= STRIP_W - KERNEL_W/2; // need some overlap on the right..

			while ( REMAIN_W > STRIP_W ) {
				// read the strip
				hipMemcpy(strip_input, DATA + DATA_H*(DATA_W-REMAIN_W-KERNEL_W/2), STRIP_SIZE,hipMemcpyHostToHost);
				// convolve the strip
				fftFunction(strip_output, strip_input, KERNEL, DATA_H, STRIP_W, KERNEL_H, KERNEL_W);
				// copy the result into the output
				hipMemcpy(output + DATA_H*(DATA_W-REMAIN_W), strip_output+DATA_H*KERNEL_W/2, STRIP_SIZE-OVERLAP_SIZE,hipMemcpyHostToHost);
				
				// set the remaining number of columns
				REMAIN_W -= STRIP_W - KERNEL_W;
			}
			// Now we have to do the remaining edge strip
			int LAST_STRIP_SIZE = REMAIN_W * DATA_H * sizeof(double);
			float* last_strip_output=(float*)malloc(LAST_STRIP_SIZE+OVERLAP_SIZE);
			float* last_strip_input=(float*)malloc(LAST_STRIP_SIZE+OVERLAP_SIZE);

			hipMemcpy(last_strip_input, DATA+DATA_H*(DATA_W-REMAIN_W-KERNEL_W/2),LAST_STRIP_SIZE+OVERLAP_SIZE,hipMemcpyHostToHost);
			fftFunction(last_strip_output, last_strip_input, KERNEL, DATA_H, REMAIN_W+KERNEL_W/2, KERNEL_H, KERNEL_W);
			hipMemcpy(output + DATA_H*(DATA_W-REMAIN_W), last_strip_output+DATA_H*KERNEL_W/2, LAST_STRIP_SIZE,hipMemcpyHostToHost);
			
			free(strip_output);
			free(strip_input);
			free(last_strip_output);
			free(last_strip_input);

		}
		else {
			fftFunction(output, DATA, KERNEL, DATA_H, DATA_W, KERNEL_H, KERNEL_W);
		}

     return output;
}

//DATA is 2D
float* sobel(float* DATA, int DATA_W, int DATA_H, float* KERNEL, int KERNEL_W, int KERNEL_H){

    float          GX[3][3];
    float         GY[3][3];
    /* 3x3 GX Sobel mask.  Ref: www.cee.hw.ac.uk/hipr/html/sobel.html */
    GX[0][0] = -1; GX[0][1] = 0; GX[0][2] = 1;
    GX[1][0] = -2; GX[1][1] = 0; GX[1][2] = 2;
    GX[2][0] = -1; GX[2][1] = 0; GX[2][2] = 1;

    /* 3x3 GY Sobel mask.  Ref: www.cee.hw.ac.uk/hipr/html/sobel.html */
    GY[0][0] =  1; GY[0][1] =  2; GY[0][2] =  1;
    GY[1][0] =  0; GY[1][1] =  0; GY[1][2] =  0;
    GY[2][0] = -1; GY[2][1] = -2; GY[2][2] = -1;

	//allocate X and Y 
	float* outputX=(float*)malloc(DATA_W*DATA_H*1*sizeof(float));
	float* outputY=(float*)malloc(DATA_W*DATA_H*1*sizeof(float));

	outputX = runFFT(DATA,DATA_W,DATA_H,*GX,3,3);
	outputX = runFFT(DATA,DATA_W,DATA_H,*GY,3,3);


	//alocate edgeImage
	float* edgeImage=(float*)malloc(DATA_W*DATA_H*1*sizeof(float));

	return edgeImage;
}

