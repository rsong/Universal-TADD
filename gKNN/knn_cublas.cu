#include "hip/hip_runtime.h"
#include "knn_cublas.cuh"

#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include <time.h>

texture<float, 2, hipReadModeElementType> texA;

__global__ void cuComputeDistanceTexture(int wA, float * B, int wB, int pB, int dim, float* AB){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if ( xIndex<wB && yIndex<wA ){
        float ssd = 0;
        for (int i=0; i<dim; i++){
            float tmp  = tex2D(texA, (float)yIndex, (float)i) - B[ i * pB + xIndex ];
            ssd += tmp * tmp;
        }
        AB[yIndex * pB + xIndex] = ssd;
    }
}


__global__ void cuInsertionSort(float *dist, int dist_pitch, int *ind, int ind_pitch, int width, int height, int k){

	// Variables
    int l, i, j;
    float *p_dist;
	int   *p_ind;
    float curr_dist, max_dist;
    int   curr_row,  max_row;
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	
    if (xIndex<width){
        
        // Pointer shift, initialization, and max value
        p_dist   = dist + xIndex;
		p_ind    = ind  + xIndex;
        max_dist = p_dist[0];
        p_ind[0] = 1;
        
        // Part 1 : sort kth firt elementZ
        for (l=1; l<k; l++){
            curr_row  = l * dist_pitch;
			curr_dist = p_dist[curr_row];
			if (curr_dist<max_dist){
                i=l-1;
				for (int a=0; a<l-1; a++){
					if (p_dist[a*dist_pitch]>curr_dist){
						i=a;
						break;
					}
				}
                for (j=l; j>i; j--){
					p_dist[j*dist_pitch] = p_dist[(j-1)*dist_pitch];
					p_ind[j*ind_pitch]   = p_ind[(j-1)*ind_pitch];
                }
				p_dist[i*dist_pitch] = curr_dist;
				p_ind[i*ind_pitch]   = l+1;
			}
			else
				p_ind[l*ind_pitch] = l+1;
			max_dist = p_dist[curr_row];
		}
        
        // Part 2 : insert element in the k-th first lines
        max_row = (k-1)*dist_pitch;
        for (l=k; l<height; l++){
			curr_dist = p_dist[l*dist_pitch];
			if (curr_dist<max_dist){
                i=k-1;
				for (int a=0; a<k-1; a++){
					if (p_dist[a*dist_pitch]>curr_dist){
						i=a;
						break;
					}
				}
                for (j=k-1; j>i; j--){
					p_dist[j*dist_pitch] = p_dist[(j-1)*dist_pitch];
					p_ind[j*ind_pitch]   = p_ind[(j-1)*ind_pitch];
                }
				p_dist[i*dist_pitch] = curr_dist;
				p_ind[i*ind_pitch]   = l+1;
                max_dist             = p_dist[max_row];
            }
        }
    }
}

__global__ void cuParallelSqrt(float *dist, int width, int pitch, int k){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (xIndex<width && yIndex<k)
        dist[yIndex*pitch + xIndex] = sqrt(dist[yIndex*pitch + xIndex]);
}


void printErrorMessage(hipError_t error, int memorySize){
    printf("==================================================\n");
    printf("MEMORY ALLOCATION ERROR  : %s\n", hipGetErrorString(error));
    printf("Whished allocated memory : %d\n", memorySize);
    printf("==================================================\n");
}

void knn(float** ref_host, int ref_width, int ref_classes, float* query_host, int query_width, int height, int k, float** dist_host, int* ind_host){
    
    unsigned int size_of_float = sizeof(float);
    unsigned int size_of_int   = sizeof(int);
    
    // Variables
    float        *query_dev;
    float        *dist_dev;
    int          *ind_dev;
    hipArray    *ref_array;
    hipError_t  result;
    size_t       query_pitch;
    size_t	     query_pitch_in_bytes;
    size_t       ind_pitch;
    size_t       ind_pitch_in_bytes;
    size_t       max_nb_query_traited;
    size_t       actual_nb_query_width;
    
    // CUDA Initialisation
    hipInit(0);
       
    // Determine maximum number of query that can be treated
    //max_nb_query_traited = ( memory_free * MAX_PART_OF_FREE_MEMORY_USED - size_of_float * ref_width*height ) / ( size_of_float * (height + ref_width) + size_of_int * k);
    max_nb_query_traited = query_width;
    
    // Allocation of global memory for query points and for distances
    result = hipMallocPitch( (void **) &query_dev, &query_pitch_in_bytes, max_nb_query_traited * size_of_float, height + ref_width);
    if (result){
        printErrorMessage(result, max_nb_query_traited*size_of_float*(height+ref_width));
        return;
    }
    query_pitch = query_pitch_in_bytes/size_of_float;
    dist_dev    = query_dev + height * query_pitch;
	

    // Allocation of global memory for indexes	
    result = hipMallocPitch( (void **) &ind_dev, &ind_pitch_in_bytes, max_nb_query_traited * size_of_int, k);
	if (result){
        hipFree(query_dev);
        printErrorMessage(result, max_nb_query_traited*size_of_int*k);
        return;
    }
    ind_pitch = ind_pitch_in_bytes/size_of_int;
    
	for(int cl=0; cl < ref_classes; cl++)
	{
			// Allocation of memory (global or texture) for reference points
				// Allocation of texture memory
				hipChannelFormatDesc channelDescA = hipCreateChannelDesc<float>();
				result = hipMallocArray( &ref_array, &channelDescA, ref_width, height );
				if (result){
					printErrorMessage(result, ref_width*height*size_of_float);
					hipFree(ind_dev);
					hipFree(query_dev);
					return;
				}
				hipMemcpyToArray( ref_array, 0, 0, ref_host[cl], ref_width * height * size_of_float, hipMemcpyHostToDevice );
        
				// Set texture parameters and bind texture to array
				texA.addressMode[0] = hipAddressModeClamp;
				texA.addressMode[1] = hipAddressModeClamp;
				texA.filterMode     = hipFilterModePoint;
				texA.normalized     = 0;
				hipBindTextureToArray(texA, ref_array);
		
			// Split queries to fit in GPU memory
			for (int i=0; i<query_width; i+=max_nb_query_traited){
        
				// Number of query points considered
				actual_nb_query_width = min( max_nb_query_traited, query_width-i );
        
				// Copy of part of query actually being treated
				hipMemcpy2D(query_dev, query_pitch_in_bytes, &query_host[i], query_width*size_of_float, actual_nb_query_width*size_of_float, height, hipMemcpyHostToDevice);
        
				// Grids ans threads
				dim3 g_16x16(actual_nb_query_width/16, ref_width/16, 1);
				dim3 t_16x16(16, 16, 1);
				if (actual_nb_query_width%16 != 0) g_16x16.x += 1;
				if (ref_width  %16 != 0) g_16x16.y += 1;
				//
				dim3 g_256x1(actual_nb_query_width/256, 1, 1);
				dim3 t_256x1(256, 1, 1);
				if (actual_nb_query_width%256 != 0) g_256x1.x += 1;
				//
				dim3 g_k_16x16(actual_nb_query_width/16, k/16, 1);
				dim3 t_k_16x16(16, 16, 1);
				if (actual_nb_query_width%16 != 0) g_k_16x16.x += 1;
				if (k  %16 != 0) g_k_16x16.y += 1;
        
				// Kernel 1: Compute all the distances
				cuComputeDistanceTexture<<<g_16x16,t_16x16>>>(ref_width, query_dev, actual_nb_query_width, query_pitch, height, dist_dev);
				    
				// Kernel 2: Sort each column
				cuInsertionSort<<<g_256x1,t_256x1>>>(dist_dev, query_pitch, ind_dev, ind_pitch, actual_nb_query_width, ref_width, k);
        
				// Kernel 3: Compute square root of k first elements
				cuParallelSqrt<<<g_k_16x16,t_k_16x16>>>(dist_dev, query_width, query_pitch, k);
        
				// Memory copy of output from device to host
				hipMemcpy2D(&dist_host[cl][i], query_width*size_of_float, dist_dev, query_pitch_in_bytes, actual_nb_query_width*size_of_float, k, hipMemcpyDeviceToHost);
				//hipMemcpy2D(&ind_host[i],  query_width*size_of_int,   ind_dev,  ind_pitch_in_bytes,   actual_nb_query_width*size_of_int,   k, hipMemcpyDeviceToHost);
			}
	}
    
    // Free memory
    hipFreeArray(ref_array);

    hipFree(ind_dev);
    hipFree(query_dev);
}


